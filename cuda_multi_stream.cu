#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hipDNN.h>
#include<hip/hip_runtime.h>
#include<hip/device_functions.h>

using namespace std;

//(A+B)/2=C
#define N (1024*1024)   //向量长度,每个流执行数据大小
#define FULL (N*20) //全部数据的大小

__global__ void kernel(int *a, int *b, int *c){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < N){
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main(){
    //查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if(!prop.deviceOverlap){
        cout << "Device will not support overlap!" << endl;
        return 0;
    }
    else{
        cout<<prop.deviceOverlap<<" yes"<<endl;
    }

    //初始化计时器时间
    hipEvent_t start, stop;
    float elapsedTime;
    //声明流和Buffer指针
    hipStream_t stream0;
    hipStream_t stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;
    //创建计时器
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //初始化流
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    //在GPU端申请内存空间
    hipMalloc((void **)&dev_a0, N * sizeof(int));
    hipMalloc((void **)&dev_b0, N * sizeof(int));
    hipMalloc((void **)&dev_c0, N * sizeof(int));
    hipMalloc((void **)&dev_a1, N * sizeof(int));
    hipMalloc((void **)&dev_b1, N * sizeof(int));
    hipMalloc((void **)&dev_c1, N * sizeof(int));
    //在CPU端申请内存空间,要使用锁页内存
    hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault);
    //初始化A,B向量
    for (int i = 0; i < FULL;i++){
        host_a[i] = rand();
        host_b[i] = rand();
    }
    //single stream开始计算
    hipEventRecord(start, 0);
    //每次传输计算长度为2*N的数据(两个流,所以是2N)
    for (int i = 0; i < FULL;i+=2*N){
        //传输数据到device,并进行计算
        hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, host_a + i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, host_b + i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
        //将计算结果从GPU传输到CPU
        hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_c + i+N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    //最后需要同步流
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Multi Time is:" << float(elapsedTime) << " s" << endl;
    //释放内存
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    return 0;
}