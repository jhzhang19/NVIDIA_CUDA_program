#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hipDNN.h>
#include<hip/hip_runtime.h>
#include<hip/device_functions.h>

using namespace std;

//(A+B)/2=C
#define N (1024*1024)   //向量长度,每个流执行数据大小
#define FULL (N*20) //全部数据的大小

__global__ void kernel(int *a, int *b, int *c){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < N){
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main(){
    //查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if(!prop.deviceOverlap){
        cout << "Device will not support overlap!" << endl;
        return 0;
    }
    else{
        cout<<prop.deviceOverlap<<" yes"<<endl;
    }

    //初始化计时器时间
    hipEvent_t start, stop;
    float elapsedTime;
    //声明流和Buffer指针
    hipStream_t stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;
    //创建计时器
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //初始化流
    hipStreamCreate(&stream);
    //在GPU端申请内存空间
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));
    //在CPU端申请内存空间,要使用锁页内存
    hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault);
    //初始化A,B向量
    for (int i = 0; i < FULL;i++){
        host_a[i] = rand();
        host_b[i] = rand();
    }
    //single stream开始计算
    hipEventRecord(start, 0);
    //每次传输计算长度为N的数据
    for (int i = 0; i < FULL;i+=N){
        //传输数据到device,并进行计算
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
        //将计算结果从GPU传输到CPU
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    //最后需要同步流
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Single Time is:" << float(elapsedTime) << " s" << endl;
    //释放内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream);

    return 0;
}