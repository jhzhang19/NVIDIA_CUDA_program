#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include<hipDNN.h>
#include<hip/hip_runtime.h>
#include<hip/device_functions.h>
// #include<opencv4/opencv2/opencv.hpp>
#include<iostream>
using namespace std;
using namespace cv;

float3 data_kernel[] = {
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
    make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
    make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
    make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
    make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),

};

int main(){
    //利用opencv的接口读取图片相关信息
    // Mat img = imread("1.jpg");
    float img = 1;
    // int imgWidth = img.cols;
    // int imgHeight = img.rows;
    // int imgChannel = img.channels();

    // Mat dst_gpu(imgHeight, imgWidth, CV_8UC3, Scalar(0, 0, 0));
    // size_t num = imgChannel * imgHeight * imgWidth * sizeof(unsigned char);
    float dst_gpu = 1;
    float num = 1;
    // 1.在gpu上分配空间
    unsigned char *in_gpu; //输入gpu的图像数据
    unsigned char *out_gpu; //输出gpu的图像数据
    float *filt_data;
    hipMalloc((void **)&filt_data, 3 * 3 * 3 * sizeof(float3));
    hipMalloc((void **)&in_gpu, num);
    hipMalloc((void **)*out_gpu, num);

    // 2.初始化句柄
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);
    
    // 3.描述tensor
    //input descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 3, imgHeight, imgWidth);
    //output descriptor
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 3, imgHeight, imgWidth);
    //kernel descriptor
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, 3, 3, 3);
    
    // 4.描述操作并设置相关参数
    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(conv_descriptor, 1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // 5.描述算法,让计算机自动选择最佳算法
    hipdnnConvolutionFwdAlgoPerf_t algo;
    cudnnGetConvolutionForwardAlgorithm_v7(handle, input_descriptor, kernel_descriptor, conv_descriptor, output_descriptor, 1, 0, &algo);


    // 6.申请工作空间
    size_t workspace_size = 0;
    //计算工作空间大小
    hipdnnGetConvolutionForwardWorkspaceSize(handle, input_descriptor, kernel_descriptor, conv_descriptor, output_descriptor, algo.algo, &workspace_size);
    //分配工作空间
    void *workspace = nullptr;
    hipMalloc(&workspace, workspace_size);

    // 7.将计算需要的数据传输到GPU
    hipMemcpy((void *)filt_data, (void *)data_kernel, 3 * 3 * 3 * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(in_gpu, img, num, hipMemcpyHostToDevice);

    // 8.开始计算
    auto alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(handle, &alpha, input_descriptor, in_gpu, 
    kernel_descriptor, filt_data, conv_descriptor, algo.algo, &workspace, workspace_size, &beta, output_descriptor, out_gpu);

    // 9.将计算结果回传到CPU
    hipMemcpy(dst_gpu, out_gpu, num, hipMemcpyDeviceToHost);

    // 10.释放资源
    hipFree(in_gpu);
    hipFree(out_gpu);
    hipFree(workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);

    hipdnnDestroy(handle);

    return 0;
}
